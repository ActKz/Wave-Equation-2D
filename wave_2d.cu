#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "wave_2d.h"
#include <time.h>
#define BLOCK_NUM 32
#define THREAD_NUM 512

extern "C"{

__global__ void kernel_cuda_update(double *olddata, double *data, double *newdata, double C, double K, double dt){

	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	//if(tid == 0)printf("len = %d\n", len);
	int x, i, j;
	int add_i, add_j, sub_i, sub_j;
	for(x = tid + bid*THREAD_NUM; x < ARR_SZ; x += THREAD_NUM*BLOCK_NUM){
		i = x / GRID_SZ;
		j = x % GRID_SZ;
		add_i = i+1 >= GRID_SZ ? i : i+1;
		add_j = j+1 >= GRID_SZ ? j : j+1;
		sub_i = i-1 < 0 ? 0 : i - 1;
		sub_j = j-1 < 0 ? 0 : j - 1;
		double pot = data[add_i * GRID_SZ + j] +
					 data[sub_i * GRID_SZ + j] +
					 data[add_j + i * GRID_SZ] +
					 data[sub_j + i * GRID_SZ] -
					 4 * data[i * GRID_SZ + j] ;
		double tmp = C * dt;
		newdata[x] = ( tmp*tmp * pot * 2 + 4 * data[x] - olddata[x] *(2 - K * dt)) / (2 + K*dt);
		//printf("X = %d, new = ")
	}


}
__global__ void cuda_move_data(double *olddata, double *data, double *newdata){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;
	for(i = tid + bid*THREAD_NUM; i < ARR_SZ; i += BLOCK_NUM*THREAD_NUM){
		olddata[i] = data[i];
		data[i] = newdata[i];
	}

}
void cuda_update(double* olddata, double* data, double* newdata,double C,double K, double dt, int step){
	double *gpu_data, *gpu_old, *gpu_new;
	hipMalloc((void**) &gpu_data, sizeof(double)*ARR_SZ);
	hipMalloc((void**) &gpu_old, sizeof(double)*ARR_SZ);
	hipMalloc((void**) &gpu_new, sizeof(double)*ARR_SZ);
	hipMemcpy(gpu_data, data, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
	hipMemcpy(gpu_old, olddata, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
	hipMemcpy(gpu_new, newdata, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
	//int _num = ARR_SZ/THREAD_NUM + 1;
	//if(_num > BLOCK_NUM) _num = BLOCK_NUM;
	int i;
	for(i = 1;i <= step; ++i){
		kernel_cuda_update<<< BLOCK_NUM, THREAD_NUM>>>(gpu_old, gpu_data, gpu_new,C, K, dt);
		cuda_move_data<<<BLOCK_NUM, THREAD_NUM>>>(gpu_old, gpu_data, gpu_new);
	}
	hipMemcpy(olddata, gpu_old, sizeof(double)*ARR_SZ, hipMemcpyDeviceToHost);
	hipMemcpy(data, gpu_data, sizeof(double)*ARR_SZ, hipMemcpyDeviceToHost);
	hipMemcpy(newdata, gpu_new, sizeof(double)*ARR_SZ, hipMemcpyDeviceToHost);
	hipFree(gpu_data);
	hipFree(gpu_old);
	hipFree(gpu_new);
}

}
