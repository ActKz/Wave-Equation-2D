#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "wave_2d.h"
#include <time.h>
#define BLOCK_NUM 32
#define THREAD_NUM 512

extern "C"{

__global__ void kernel_cuda_update(double *olddata, double *data, double *newdata, double C, double K, double dt, int step){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int x, i, j, y;
    int add_i, add_j, sub_i, sub_j;
        for(x = tid + bid*THREAD_NUM; x < ARR_SZ; x += THREAD_NUM*BLOCK_NUM){
        	i = x / GRID_SZ;
        	j = x % GRID_SZ;
            add_i = i+1 >= GRID_SZ ? i : i+1;
            add_j = j+1 >= GRID_SZ ? j : j+1;
        	sub_i = i-1 < 0 ? 0 : i - 1;
        	sub_j = j-1 < 0 ? 0 : j - 1;
        	double pot = data[add_i * GRID_SZ + j] +
        				 data[sub_i * GRID_SZ + j] +
        				 data[add_j + i * GRID_SZ] +
        				 data[sub_j + i * GRID_SZ] -
        				 4 * data[i * GRID_SZ + j] ;
        	double tmp = C * dt;
        	newdata[x] = ( tmp*tmp * pot * 2 + 4 * data[x] - olddata[x] *(2 - K * dt)) / (2 + K*dt);
        }
}

void cuda_update(double* olddata, double* data, double* newdata,double C,double K, double dt, int step){
    double *gpu_data, *gpu_old, *gpu_new, *tmp;
    hipMalloc((void**) &gpu_data, sizeof(double)*ARR_SZ);
    hipMalloc((void**) &gpu_old, sizeof(double)*ARR_SZ);
    hipMalloc((void**) &gpu_new, sizeof(double)*ARR_SZ);
    hipMemcpy(gpu_data, data, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
    hipMemcpy(gpu_old, olddata, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
    hipMemcpy(gpu_new, newdata, sizeof(double)*ARR_SZ, hipMemcpyHostToDevice);
    int i;
    for(i = 1;i <= step; ++i){
    	kernel_cuda_update<<< BLOCK_NUM, THREAD_NUM>>>(gpu_old, gpu_data, gpu_new,C, K, dt, step);
        tmp = gpu_old;
        gpu_old = gpu_data;
        gpu_data = gpu_new;
        gpu_new = tmp;
    }
    hipMemcpy(data, gpu_data, sizeof(double)*ARR_SZ, hipMemcpyDeviceToHost);
    hipFree(gpu_data);
    hipFree(gpu_old);
    hipFree(gpu_new);
}

}
